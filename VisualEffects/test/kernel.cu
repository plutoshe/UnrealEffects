#include "hip/hip_runtime.h"
#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1
#include <stdio.h>

__global__ void hello()
{
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}
void hello1()
{
	hello << <NUM_BLOCKS, BLOCK_WIDTH >> > ();
	hipDeviceSynchronize();
}
