#include "hip/hip_runtime.h"
#include "Compution.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <qdebug.h>
#include <cmath>
//
__device__ int clamp(float x, float a, float b)
{
	
    return max(a, min(b, x));
}

__device__ int get1Dpos(int i, int j, int delta)
{
    return i * delta + j;
}

__global__
void Advect(int i_n,
	int i_m,
	float i_deltaTime,
    const float* const i_ux,
    const float* const i_uy,
    float* o_ux,
    float* o_uy)
{
    int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
    int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
	float bx = i - i_deltaTime * i_ux[get1Dpos(i, j, i_m + 2)] * i_n;
	float by = j - i_deltaTime * i_uy[get1Dpos(i, j, i_m + 2)] * i_m;
	bx = clamp(bx, 0.5, i_n + 0.5);
	by = clamp(by, 0.5, i_m + 0.5);

	int bi0 = (int)bx; int bj0 = by;
	int bi1 = bi0 + 1; int bj1 = bj0 + 1;
	float s1 = bx - bi0; float t1 = by - bj0;
	float s0 = 1 - s1; float t0 = 1 - t1;
	o_ux[get1Dpos(i, j, i_m + 2)] =
		s0 * (t0 * i_ux[get1Dpos(bi0, bj0, i_m + 2)] + t1 * i_ux[get1Dpos(bi0, bj1, i_m + 2)]) +
		s1 * (t0 * i_ux[get1Dpos(bi1, bj0, i_m + 2)] + t1 * i_ux[get1Dpos(bi1, bj1, i_m + 2)]);
	o_uy[get1Dpos(i, j, i_m + 2)] =
		s0 * (t0 * i_uy[get1Dpos(bi0, bj0, i_m + 2)] + t1 * i_uy[get1Dpos(bi0, bj1, i_m + 2)]) +
		s1 * (t0 * i_uy[get1Dpos(bi1, bj0, i_m + 2)] + t1 * i_uy[get1Dpos(bi1, bj1, i_m + 2)]);;
}

__global__ void test(int i_n, int i_m, const float* const i_in, float* i_out)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
	int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
    
    i_out[get1Dpos(i, j, i_m + 2)] = i_in[get1Dpos(i, j, i_m + 2)] + 10;
	
}

__global__ void AddForce(int i_n, int i_m, float2 i_forceOrigin, float i_forceExponennt, float2 i_forceVector, const float* const i_ux, const float* const i_uy, float* o_ux, float* o_uy)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
	int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
	float sx =  i_forceOrigin.x - i * 1.0 / i_n;
	float sy = i_forceOrigin.y - j * 1.0 / i_m;
	float amp = exp(-i_forceExponennt*sqrt(sx*sx + sy * sy));

	o_ux[get1Dpos(i, j, i_m + 2)] = i_ux[get1Dpos(i, j, i_m + 2)] + i_forceVector.x * amp;
	o_uy[get1Dpos(i, j, i_m + 2)] = i_uy[get1Dpos(i, j, i_m + 2)] + i_forceVector.y * amp;
}


__global__ void Diffuse(int i_n, int i_m, float i_alpha, float i_beta, const float* const i_origin, const float* const i_grid, float* o_grid)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
	int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
	o_grid[get1Dpos(i, j, i_m + 2)] =
		(i_origin[get1Dpos(i, j, i_m + 2)] * i_alpha +
			i_grid[get1Dpos(i - 1, j, i_m + 2)] +
			i_grid[get1Dpos(i, j - 1, i_m + 2)] +
			i_grid[get1Dpos(i + 1, j, i_m + 2)] +
			i_grid[get1Dpos(i, j + 1, i_m + 2)]) * i_beta;
}



__global__ void ProjectStart(int i_n, int i_m, float i_h, const float* const i_ux, const float* const i_uy, float* o_div, float* o_p)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
	int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
	o_div[get1Dpos(i, j, i_m + 2)] =
		0.5f * i_h * (
			i_ux[get1Dpos(i + 1, j, i_m + 2)] -
			i_ux[get1Dpos(i - 1, j, i_m + 2)] +
			i_uy[get1Dpos(i, j + 1, i_m + 2)] -
			i_uy[get1Dpos(i, j - 1, i_m + 2)]);
	o_p[get1Dpos(i, j, i_m + 2)] = 0;

	/*SetBoundry(i_n, i_m, o_div);
	SetBoundry(i_n, i_m, o_p);*/

}

__global__ void ProjectFinish(int i_n, int i_m, float i_h, const float* const i_ux, const float* const i_uy, const float* const i_p, float* o_ux, float* o_uy)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n);
	int j = clamp(blockIdx.y * blockDim.y + threadIdx.y + 1, 1, i_m);
	float p1 = i_p[get1Dpos((i <= 2? 1 : i - 1), j, i_m + 2)];
	float p2 = i_p[get1Dpos((i < i_n - 1) ? i+ 1 : i_n, j, i_m + 2)];
	float p3 = i_p[get1Dpos(i, j <= 2?1: j - 1, i_m + 2)];
	float p4 = i_p[get1Dpos(i, (j < i_m - 1)?j + 1: i_m, i_m + 2)];
	float ux = i_ux[get1Dpos(i, j, i_m + 2)] - 0.5 * (p2 - p1) / i_h;
	float uy = i_uy[get1Dpos(i, j, i_m + 2)] - 0.5 * (p4 - p3) / i_h;
	o_ux[get1Dpos(i, j, i_m + 2)] = ux;
	o_uy[get1Dpos(i, j, i_m + 2)] = uy;
	if (i == 1)
	{
		o_ux[get1Dpos(0, j, i_m + 2)] = -ux;
		o_uy[get1Dpos(0, j, i_m + 2)] = uy;
	}
	if (i == i_n)
	{
		o_ux[get1Dpos(0, j, i_m + 2)] = -ux;
		o_uy[get1Dpos(0, j, i_m + 2)] = uy;
	}
	if (j == 1)
	{
		o_ux[get1Dpos(0, j, i_m + 2)] = ux;
		o_uy[get1Dpos(0, j, i_m + 2)] = -uy;
	}
	if (j == i_m)
	{
		o_ux[get1Dpos(0, j, i_m + 2)] = ux;
		o_uy[get1Dpos(0, j, i_m + 2)] = -uy;
	}
	

	//SetBoundry(i_n, i_m, o_v);
	
	//o_v[get1Dpos(0, 0, i_m + 2)] = 0.5 * (o_v[get1Dpos(1, 0, i_m + 2)] + o_v[get1Dpos(0, 1, i_m + 2)]);
	//o_v[get1Dpos(0, i_m + 1, i_m + 2)] = 0.5 * (o_v[get1Dpos(1, i_m + 1, i_m + 2)] + o_v[get1Dpos(0, i_m, i_m + 2)]);
	//o_v[get1Dpos(i_n + 1, 0, i_m + 2)] = 0.5 * (o_v[get1Dpos(i_n, 0, i_m + 2)] + o_v[get1Dpos(i_n + 1, 1, i_m + 2)]);
	//o_v[get1Dpos(i_n + 1, i_m + 1, i_m + 2)] = 0.5 * (o_v[get1Dpos(i_n, i_m + 1, i_m + 2)] + o_v[get1Dpos(i_n + 1, i_m, i_m + 2)]);
}


__global__ void SetBoundry(int i_n, int i_m, float* o_v, int i_status1, int i_status2)
{
	int i = clamp(blockIdx.x * blockDim.x + threadIdx.x + 1, 1, i_n + i_m + 1);
	//int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
	if (i <= i_m)
	{ 
	
		o_v[get1Dpos(0, i, i_m + 2)] = i_status1 * o_v[get1Dpos(1, i, i_m + 2)];
		o_v[get1Dpos(i_n + 1, i, i_m + 2)] = i_status1 * o_v[get1Dpos(i_n, i, i_m + 2)];
	}
	else
	{
		i -= i_m;
		o_v[get1Dpos(i, 0, i_m + 2)] = i_status2 * o_v[get1Dpos(i, 1, i_m + 2)];
		o_v[get1Dpos(i, i_m + 1, i_m + 2)] = i_status2 * o_v[get1Dpos(i, i_m, i_m + 2)];
	}
	
}

void Initialization()
{

}

void CalculateNewVelocity(const int i_n, const int i_m, float* o_ux, float* o_uy, float i_deltaTime, float i_vicosityParam, float i_forceExponentParam, QVector2D i_mousePosition, QVector2D i_forceVector, bool i_gpu)
{
	//m_interVelocity1 = m_velocity;
	//m_interVelocity2 = m_velocity;
	
	if (i_deltaTime < -1e7)
	{
		i_deltaTime = 1e-7;
	}
	double paramB = i_deltaTime * i_vicosityParam;
	if (paramB == 0)
	{
		paramB = 0.00000000001f;
	}
	// initialization
	
	const dim3 blockSize((i_n - 1) / 32 + 1, (i_m - 1) / 32 + 1, 1);
	const dim3 gridSize(32, 32, 1);
	const dim3 borderBlockSize((i_n + i_m) / 1024);
	const dim3 borderGridSize(1024, 1, 1);
	float* ux1 = nullptr;
	float* uy1 = nullptr;
	float* ux2 = nullptr;
	float* uy2 = nullptr;
	float* ux3 = nullptr;
	float* uy3 = nullptr;
	float* p1 = nullptr;
	float* p2 = nullptr;
	float* div = nullptr;
	int totalNum = (i_n + 2) * (i_m + 2);
	int totalSize = sizeof(float) * (i_n + 2) * (i_m + 2);
	if (i_gpu)
	{
		hipMalloc(&ux1, totalSize);
		hipMalloc(&ux2, totalSize);
		hipMalloc(&uy1, totalSize);
		hipMalloc(&uy2, totalSize);
		hipMalloc(&ux3, totalSize);
		hipMalloc(&uy3, totalSize);
		hipMalloc(&p1, totalSize);
		hipMalloc(&p2, totalSize);
		hipMalloc(&div, totalSize);


		hipMemcpy(ux1, o_ux, totalSize, hipMemcpyHostToDevice);
		hipMemcpy(uy1, o_uy, totalSize, hipMemcpyHostToDevice);
	}
	else
	{
		ux1 = new float[totalNum];
		uy1 = new float[totalNum];
		ux2 = new float[totalNum];
		uy2 = new float[totalNum];
		ux3 = new float[totalNum];
		uy3 = new float[totalNum];
		p1 = new float[totalNum];
		p2 = new float[totalNum];
		div = new float[totalNum];
		hipMemset(div, 0, totalSize * sizeof(float));
		hipMemcpy(ux1, o_ux, totalSize, hipMemcpyHostToHost);
		hipMemcpy(uy1, o_uy, totalSize, hipMemcpyHostToHost);

	}

	double dx = 1.0 / i_n;
	double dif_alpha = (dx * dx) / (paramB);
	if (i_gpu)
	{ 
		Advect << < gridSize, blockSize >> > (i_n, i_m, i_deltaTime, ux1, uy1, ux2, uy2);
		hipMemcpy(ux1, ux2, totalSize, hipMemcpyDeviceToDevice);
		hipMemcpy(uy1, uy2, totalSize, hipMemcpyDeviceToDevice);
	}
	else
	{
		Fluid::Computation::Advect(i_n, i_m, i_deltaTime, ux1, uy1, ux2, uy2);
		hipMemcpy(ux1, ux2, totalSize, hipMemcpyHostToHost);
		hipMemcpy(uy1, uy2, totalSize, hipMemcpyHostToHost);

	}
	

	for (int i = 0; i < 20; i++)
	{
		if (i_gpu)
		{
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, dif_alpha, 1 / (4 + dif_alpha), ux1, ux2, ux3);
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, dif_alpha, 1 / (4 + dif_alpha), uy1, uy2, uy3);
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, ux3, -1, 1);
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, uy3, 1, -1);
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, dif_alpha, 1 / (4 + dif_alpha), ux1, ux3, ux2);
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, dif_alpha, 1 / (4 + dif_alpha) , uy1, uy3, uy2);
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, ux2, -1, 1);
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, uy2, 1, -1);
		}
		else
		{
			Fluid::Computation::Diffuse(i_n, i_m, dif_alpha, 1.0 / (4.0 + dif_alpha), ux1, ux2, ux3);
			Fluid::Computation::Diffuse(i_n, i_m, dif_alpha, 1.0 / (4.0 + dif_alpha), uy1, uy2, uy3);
			Fluid::Computation::SetBoundry(i_n, i_m, ux3, 1, false);
			Fluid::Computation::SetBoundry(i_n, i_m, uy3, 2, false);
			Fluid::Computation::Diffuse(i_n, i_m, dif_alpha, 1.0 / (4.0 + dif_alpha), ux1, ux3, ux2);
			Fluid::Computation::Diffuse(i_n, i_m, dif_alpha, 1.0 / (4.0 + dif_alpha), uy1, uy3, uy2);
			Fluid::Computation::SetBoundry(i_n, i_m, ux2, 1, false);
			Fluid::Computation::SetBoundry(i_n, i_m, uy2, 2, false);
			//Fluid::Computation::SetBoundry(m_height, m_width, m_interVelocity1);
		}
	}
	
	if (i_gpu)
	{
		AddForce << <gridSize, blockSize >> > (i_n, i_m, make_float2(i_mousePosition.x(), i_mousePosition.y()), i_forceExponentParam, make_float2(i_forceVector.x(), i_forceVector.y()), ux2, uy2, ux3, uy3);
		ProjectStart << <gridSize, blockSize >> > (i_n, i_m, 1.0 / i_n, ux3, uy3, div, p1);
		SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, div, 1, 1);
		SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, p1, 1, 1);
	}
	else
	{
		Fluid::Computation::AddForce(i_n, i_m, i_mousePosition, i_forceExponentParam, i_forceVector, ux2, uy2, ux3, uy3);
		Fluid::Computation::ProjectStart(i_n, i_m, 0.5 * dx, ux3, uy3, div, p1);
		Fluid::Computation::SetBoundry(i_n, i_m, div, 0, true);
		Fluid::Computation::SetBoundry(i_n, i_m, p1, 0, true);
	}
	
	//	
	

	for (int i = 0; i < 20; i++)
	{
		if (i_gpu)
		{
		
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, 1.0, 1.0/4, div, p1, p2);
			
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, p2, 1, 1);
			Diffuse << <gridSize, blockSize >> > (i_n, i_m, 1.0, 1.0 / 4, div, p2, p1);
			SetBoundry << <borderGridSize, borderBlockSize >> > (i_n, i_m, p1, 1, 1);
		}
		else
		{
			Fluid::Computation::Diffuse(i_n, i_m, -1, 1.0 / 4, div, p1, p2);
			Fluid::Computation::SetBoundry(i_n, i_m, p2, 0, true);
			//Fluid::Compution::SetBoundry(m_height, m_width, m_P2);
			Fluid::Computation::Diffuse(i_n, i_m, -1, 1.0 / 4, div, p2, p1);
			Fluid::Computation::SetBoundry(i_n, i_m, p1, 0, true);
			//Fluid::Compution::SetBoundry(m_height, m_width, m_P1);
			qDebug() << p1[0] << p2[0];
		}
	}
	if (i_gpu)
	{
		ProjectFinish << <gridSize, blockSize >> > (i_n, i_m, dx, ux3, uy3, p1, ux1, uy1);
		hipMemcpy(o_ux, ux1, totalSize, hipMemcpyDeviceToHost);
		hipMemcpy(o_uy, uy1, totalSize, hipMemcpyDeviceToHost);
		Fluid::Computation::SetBoundry(i_n, i_m, o_ux, 1, true);
		Fluid::Computation::SetBoundry(i_n, i_m, o_uy, 2, true);
	}
	else
	{
		Fluid::Computation::ProjectFinish(i_n, i_m, 0.5 * dx, ux3, uy3, p1, ux1, uy1);
		hipMemcpy(o_ux, ux1, totalSize, hipMemcpyHostToHost);
		hipMemcpy(o_uy, uy1, totalSize, hipMemcpyHostToHost);
		Fluid::Computation::SetBoundry(i_n, i_m, o_ux, 1, true);
		Fluid::Computation::SetBoundry(i_n, i_m, o_uy, 2, true);
	}
}

void test1(int i_n, int i_m)
{
	int m_height = 100;
	int m_width = 100;
	
	float* result = new float[sizeof(float) * (m_height + 2) * (m_width + 2)];
	const dim3 blockSize((m_height - 1) / 32 + 1, (m_width - 1) / 32 + 1, 1);
	const dim3 gridSize(32, 32, 1);
	float* d_a = nullptr;
	float* d_b = nullptr;
	
	auto a = hipMalloc(&d_a, sizeof(float) * (m_height + 2) * (m_width + 2));
	auto b = hipMalloc(&d_b, sizeof(float) * (m_height + 2) * (m_width + 2));
	hipMemset(d_a, 0, sizeof(float) * (m_height + 2) * (m_width + 2));
	hipMemset(d_b, 0, sizeof(float) * (m_height + 2) * (m_width + 2));
	hipMemcpy(result, d_a, sizeof(float) * (m_height + 2) * (m_width + 2), hipMemcpyDeviceToHost);
	//hipMemcpy(d_b, result, sizeof(float) * (m_height + 2) * (m_width + 2), hipMemcpyHostToDevice);
	
	test<<<gridSize, blockSize>>>(m_height, m_width, d_a, d_b);
	hipMemcpy(result, d_b, sizeof(float) * (m_height + 2) * (m_width + 2), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	for (int i = 1; i <= 100; i++ )
		for (int j = 1; j <= 100; j++)
			if (result[i * 102 + 100] != 10)
			{
			
			}
}
