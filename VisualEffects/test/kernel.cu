#include "hip/hip_runtime.h"
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
	printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}

int main()
{
	hello << <NUM_BLOCKS, BLOCK_WIDTH >> > ();
	hipDeviceSynchronize();
	printf("That's all!");
	return 0;
}